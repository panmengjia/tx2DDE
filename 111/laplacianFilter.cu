#include "hip/hip_runtime.h"
//
// CUDA implementation of Laplacian Filter
//
#include "opencv2/opencv.hpp"
#include <opencv2/highgui.hpp>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
//#include <qdebug.h>

#define BLOCK_SIZE      8
#define FILTER_WIDTH    15
#define FILTER_HEIGHT   15

using namespace std;

const int inputSize = sizeof(uchar)*1920 * 1080;
const int outputSize = sizeof(uchar)*1920 * 1080;
const int kernelSize = sizeof(float)*FILTER_WIDTH * FILTER_HEIGHT;
unsigned char *d_input, *d_output;
float *d_kernel;


bool initialized=false;

// Run Laplacian Filter on GPU
__global__ void laplacianFilter(unsigned char *srcImage, unsigned char *dstImage, unsigned int width, unsigned int height, float * kernel0)
{
   int x = blockIdx.x*blockDim.x + threadIdx.x;
   int y = blockIdx.y*blockDim.y + threadIdx.y;

   //float kernel[3][3] = {-1, -1, -1, -1, 8, -1, -1, -1, -1};
   // only threads inside image will write results
   if((x>=FILTER_WIDTH/2) && (x<(width-FILTER_WIDTH/2)) && (y>=FILTER_HEIGHT/2) && (y<(height-FILTER_HEIGHT/2)))
   {
         // Sum of pixel values
         float sum = 0;
         // Loop inside the filter to average pixel values
         for(int ky=-FILTER_HEIGHT/2; ky<=FILTER_HEIGHT/2; ky++) {
            for(int kx=-FILTER_WIDTH/2; kx<=FILTER_WIDTH/2; kx++) {
               float fl = srcImage[((y+ky)*width + (x+kx))];
               sum += fl*kernel0[(ky+FILTER_HEIGHT/2)*FILTER_WIDTH + kx+FILTER_WIDTH/2];
            }
         }
         dstImage[(y*width+x)] =  sum;
   }
}


// The wrapper to call laplacian filter
extern "C" void laplacianFilter_GPU_wrapper(const cv::Mat& input, cv::Mat& output, const cv::Mat& kernel)
{
        // Use cuda event to catch time
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        if (!initialized)
        {
            cout<<"111"<<endl;
            // Allocate device memory
            hipMalloc<unsigned char>(&d_input,inputSize);
            hipMalloc<unsigned char>(&d_output,outputSize);
            hipMalloc<float>(&d_kernel,kernelSize);

            initialized=true;
        }

        // Copy data from OpenCV input image to device memory
        hipMemcpy(d_input,input.ptr(),inputSize,hipMemcpyHostToDevice);
//        cout<<"22222"<<endl;
        hipMemcpy(d_kernel,kernel.ptr(),kernelSize,hipMemcpyHostToDevice);

        // Specify block size
        const dim3 block(BLOCK_SIZE,BLOCK_SIZE);

        // Calculate grid size to cover the whole image
        const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);

        // Start time
        hipEventRecord(start,0);

        laplacianFilter<<<grid,block>>>(d_input, d_output, output.cols, output.rows, d_kernel);

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        printf("Kernel time: %.2f ms\n", milliseconds);

        // Stop time

        //Copy data from device memory to output image



        hipMemcpy(output.ptr(),d_output,outputSize,hipMemcpyDeviceToHost);

        //Free the device memory
//        hipFree(d_input);
//        hipFree(d_output);
//        hipFree(d_kernel);

//        cout<<"33333333333333333333"<<endl;


}


